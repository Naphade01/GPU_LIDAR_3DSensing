#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudaImageTransformation.h"


#define transMatCols    4
#define transMatRows    4

static float *d_transformationMat = NULL;
static twoDCoordinates *d_transformedCoordinates = NULL;

__device__ twoDCoordinates
cudaTransform(float* transformationMat,int xValue, int yValue,
              const int planeDistance,const twoDCoordinates offset,
              const int projectionPlane)
{
    twoDCoordinates twoDPoint;
    float matrixResult[transMatRows];
    
    float threeDPoint[transMatRows];
    
    switch(projectionPlane)
    {
        case 1:
            threeDPoint[0] = (float)planeDistance;
            threeDPoint[1] = (float)xValue;
            threeDPoint[2] = (float)yValue;
            break;
        case 2:
            threeDPoint[0] = (float)xValue;
            threeDPoint[1] = (float)planeDistance;
            threeDPoint[2] = (float)yValue;
            break;
        case 3:
            threeDPoint[0] = (float)xValue;
            threeDPoint[1] = (float)yValue;
            threeDPoint[2] = (float)planeDistance;
            break;
        default:
            //Error handling to be performed
            break;
    }
    
    float sum = 0;
    
    for(int i=0;i<transMatRows;i++)
    {
        for(int j=0;j<transMatCols;j++)
        {
            sum += (transformationMat[(i*transMatCols)+j] * threeDPoint[j]);
        }
        matrixResult[i] = sum;
        sum = 0.0;
    }
    
    twoDPoint.x = (int)((float)planeDistance * matrixResult[0]/matrixResult[2]);

    twoDPoint.x += offset.x;
                                  
    twoDPoint.y = (int)((float)planeDistance * matrixResult[1]/matrixResult[2]);
    twoDPoint.y += offset.y;
                                  
    return twoDPoint;
}

__global__ void
cudaGetTransformedCoordinates(twoDCoordinates* transformedCoordinates,
                              float* transformationMat,
                              const int planeDistance,
                              const twoDCoordinates offset, 
                              const int projectionPlane,
                              const int imageWidth,
                              const int imageHeight)
{
    //int2 temp;
    int xTid = (blockIdx.x + blockDim.x) + threadIdx.x;
    int yTid = (blockIdx.y + blockDim.y) + threadIdx.y;
    
    if((xTid < imageWidth)&& (yTid <imageHeight))
    {
//        temp 
        transformedCoordinates[(xTid*imageWidth)+yTid]= cudaTransform(transformationMat,
                                                             xTid,
                                                             yTid,
                                                             planeDistance,
                                                             offset,
                                                             projectionPlane);
                             
  //  transformedCoordinates[(xTid*imageWidth)+yTid].x = temp.x;
  //  transformedCoordinates[xTid][yTid].y = temp.y;                                                               
    }
}

void 
apiCudaGetTransformedCoordinates(twoDCoordinates* h_transformedCoordinates,
                                 float* h_transformationMat,
                                 const int planeDistance,
                                 const int xOffset,
                                 const int yOffset,
                                 const int projectionPlane,
                                 const int imageWidth,
                                 const int imageHeight)
{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    twoDCoordinates offset;
    offset.x = xOffset;
    offset.y = yOffset;
    
    unsigned long int transformedCoordinatesSize = imageWidth*imageHeight*sizeof(twoDCoordinates);
    
    unsigned long int transformationMatSize = transMatCols*transMatRows*sizeof(float);
    
    if(d_transformedCoordinates==NULL)
    {
        err = hipMalloc(&d_transformedCoordinates, transformedCoordinatesSize);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device memory for Transformed Coordinates Array"
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    
    err = hipMemset(d_transformedCoordinates,0,transformedCoordinatesSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to initialize device memory for Transformed Coordinates Array"
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if(d_transformationMat == NULL)
    {
        err = hipMalloc(&d_transformationMat, transformationMatSize);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device memory for Transformation Matrix"
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    
    
        //Copy host data on device
        err = hipMemcpy(d_transformationMat,h_transformationMat,transformationMatSize,hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy transformation Matrix from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    
    //Kernel call   
    //***Hard coded for 320x240 image... need to generalize it
    dim3 blocks(10,10);
    dim3 threadsPerBlock(32,24);
    
    cudaGetTransformedCoordinates<<<blocks,threadsPerBlock>>>(d_transformedCoordinates,
                                                              d_transformationMat,
                                                              planeDistance,
                                                              offset, 
                                                              projectionPlane,
                                                              imageWidth,
                                                              imageHeight);
                                                              
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch cudaGetTransformedCoordinates kernel"
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    hipDeviceSynchronize();
    
    err = hipMemcpy(h_transformedCoordinates,d_transformedCoordinates,transformedCoordinatesSize,hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy transformed Coordinates from device to host"
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

//Need to handle gracefull exit
    
void freeDeviceAllocatedMemory()
{
    hipFree(d_transformationMat);
    hipFree(d_transformedCoordinates);
}
