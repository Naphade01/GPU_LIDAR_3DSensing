#include "hip/hip_runtime.h"
#include "cudaImageTransform.hpp"
#include <opencv2/core/cuda_types.hpp>
#include <stdio.h>


using namespace std;
using namespace cv;
using namespace cv::cuda;

static unsigned char *d_srcImage=NULL;
static unsigned char *d_destImage=NULL;
static float *d_transMat=NULL;

#define transMatCols    4
#define transMatRows    4

__device__ twoDCoordinates
cudaTransform(const float *transMat,int xValue, int yValue,const twoDCoordinates offset,
              const int destImageHeight,const int constScreenCoordinate,
              const int projectionPlane,const int planeDistance)
{

    //printf("xTid:%d yTid:%d TransMatStep:%d\n",xValue,yValue,transMatStep);
    
    //printf("Const Screen Coordinate: %d\n", constScreenCoordinate);
    
    twoDCoordinates twoDPoint;
    float matrixResult[transMatRows];
    
    float threeDPoint[transMatRows];
    
    threeDPoint[3] = 1;
    
    switch(projectionPlane)
    {
        case 1:
            threeDPoint[0] = (float)constScreenCoordinate;
            threeDPoint[1] = (float)xValue;
            threeDPoint[2] = (float)yValue;
            break;
        case 2:
            threeDPoint[0] = (float)xValue;
            threeDPoint[1] = (float)constScreenCoordinate;
            threeDPoint[2] = (float)yValue;
            break;
        case 3:
            threeDPoint[0] = (float)xValue;
            threeDPoint[1] = (float)yValue;
            threeDPoint[2] = (float)constScreenCoordinate;
            break;
        default:
            //Error handling to be performed
            break;
    }
    
    int transMatLoc=0;
    
    float sum = 0;
    
    for(int i=0;i<transMatRows;i++)
    {
        for(int j=0;j<transMatCols;j++)
        {
            transMatLoc = i*transMatCols + j;
            
            //printf("%f ",transMat[transMatLoc]);
            
            sum += (transMat[transMatLoc] * threeDPoint[j]);
        }
        //printf("\n");
        matrixResult[i] = sum;
        sum = 0.0;
    }
    
    twoDPoint.x = (int)((float)planeDistance * matrixResult[0]/matrixResult[2]);

    twoDPoint.x += offset.x;
                                  
    twoDPoint.y = (int)((float)planeDistance * matrixResult[1]/matrixResult[2]);
    twoDPoint.y += offset.y;
    
    //twoDPoint.y = destImageHeight-twoDPoint.y;
            
    return twoDPoint;
}

__global__ void
cudaGetTransformedCoordinates(unsigned char *src, 
                              unsigned char *dest,
                              const float* transMat,  
                              const int srcImageStep,
                              const int destImageStep,                          
                              const twoDCoordinates offset,
                              const int constScreenCoordinate, 
                              const int projectionPlane,
                              const int planeDistance,
                              const int srcImageWidth,
                              const int srcImageHeight,
                              const int destImageHeight,
                              const int srcImageBytes,
                              const int destImageBytes,
                              const twoDCoordinates threadLimits)
{
    //int2 temp;
    int xTid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int yTid = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    //Location of colored pixel in output
    
    long int destColorLoc=0;
    long int srcColorLoc =0;
    
    twoDCoordinates transCoordinates;
    
    if((xTid < threadLimits.x) && (yTid < threadLimits.y))
    {
        srcColorLoc = (yTid * srcImageStep) + (3 * xTid);
        
        transCoordinates = cudaTransform(transMat,xTid,yTid,offset,destImageHeight,
                                         constScreenCoordinate, projectionPlane,
                                         planeDistance);
        
           
        destColorLoc = (transCoordinates.y * destImageStep) + (3*transCoordinates.x);
        
        if((srcColorLoc < (srcImageBytes-2))&&(destColorLoc < (destImageBytes-2)))
        { 
            if((xTid == 0) && (yTid==0))
            {
                printf("xTid:%d yTid:%d Tx:%d Ty:%d destColorLoc:%d srcLoc:%d\n",xTid,yTid,transCoordinates.x,transCoordinates.y,destColorLoc,srcColorLoc);
            }
            
            dest[destColorLoc]=src[srcColorLoc];
            dest[destColorLoc+1]=src[srcColorLoc+1];
            dest[destColorLoc+2]=src[srcColorLoc+2];
        }
    }
}


void 
cudaImageProjectioncaller(const cv::Mat& h_srcImage, 
                          cv::Mat& h_destImage,
                          const cv::Mat& h_transMat,
                          const int xOffset,
                          const int yOffset,
                          cv::Point3d screenCoordinates,
                          const int projPlane,
                          const int projPlaneDist)
{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    twoDCoordinates offset;
    offset.x = xOffset;
    offset.y = yOffset;
       
    const int srcImageBytes = h_srcImage.step * h_srcImage.rows;
    const int destImageBytes = h_destImage.step * h_destImage.rows;
    const int transMatBytes = h_transMat.step * h_transMat.rows;
    
    cout << "Offset :" << offset.x << " "<<offset.y << endl;
    
    cout << "Plane Distance: " << projPlaneDist << endl;
    
    /*static float* transMatPtr = NULL;
    if (transMatPtr == NULL)
    {
        transMatPtr = (float*)malloc(transMatBytes);
        memcpy(transMatPtr, h_transMat.ptr(),transMatBytes);
    }
    
    for(int i =0; i<h_transMat.rows; i++ )
    {
        for(int j =0; j<h_transMat.cols; j++ )
            cout << transMatPtr[i*h_transMat.cols+j] << " ";
        
        cout << endl;
    }*/
    
        
    if(d_srcImage==NULL)
    {
        err = hipMalloc(&d_srcImage, srcImageBytes);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device memory for Source Image "
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    //Copy host data on device
    err = hipMemcpy(d_srcImage,h_srcImage.ptr(),srcImageBytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy Source Image from host to device "
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    

    if(d_transMat==NULL)
    {
        err = hipMalloc(&d_transMat, transMatBytes);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device memory for Trans Matrix "
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        err = hipMemcpy(d_transMat,h_transMat.ptr(),transMatBytes,hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy Trans Matrix from host to device "
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    
    if(d_destImage==NULL)
    {
        err = hipMalloc(&d_destImage, destImageBytes);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device memory for Source Image"
                            "(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    //Copy host data on device
    err = hipMemcpy(d_destImage,h_destImage.ptr(),destImageBytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy Output Image from host to device "
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    //Kernel call   
    dim3 blocks(10,10);
    dim3 threadsPerBlock(0,0);
    int constScreenCoordinate;
    twoDCoordinates threadLimits;
    
    switch(projPlane)
    {
        case 1:
            constScreenCoordinate = screenCoordinates.x;
            threadsPerBlock.x = screenCoordinates.y/10;
            threadsPerBlock.y = screenCoordinates.z/10;
            threadLimits.x    = screenCoordinates.y;
            threadLimits.y    = screenCoordinates.z;
            break;
        case 2:
            constScreenCoordinate = screenCoordinates.y;
            threadsPerBlock.x = screenCoordinates.x/10;
            threadsPerBlock.y = screenCoordinates.z/10;
            threadLimits.x    = screenCoordinates.x;
            threadLimits.y    = screenCoordinates.z;
            break;
        case 3:
            constScreenCoordinate = screenCoordinates.z;
            threadsPerBlock.x = screenCoordinates.x/10;
            threadsPerBlock.y = screenCoordinates.y/10;
            threadLimits.x    = screenCoordinates.x;
            threadLimits.y    = screenCoordinates.y;            
            break;
        default:
            //Error handling to be performed
            break;
    }
    
    cudaGetTransformedCoordinates<<<blocks,threadsPerBlock>>>(d_srcImage,
                                                              d_destImage,
                                                              d_transMat,
                                                              h_srcImage.step,
                                                              h_destImage.step,
                                                              offset,
                                                              constScreenCoordinate,
                                                              projPlane,
                                                              projPlaneDist,
                                                              h_srcImage.cols,
                                                              h_srcImage.rows,
                                                              h_destImage.rows,
                                                              srcImageBytes,
                                                              destImageBytes,
                                                              threadLimits);
                                                              
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch cudaGetTransformedCoordinates kernel"
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
                                                              
    hipDeviceSynchronize();
    
    err = hipMemcpy(h_destImage.ptr(),d_destImage,destImageBytes,hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy Output Image from device to host"
                        "(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


void freeDeviceAllocatedMemory(int sig)
{
    hipFree(d_srcImage);
    hipFree(d_transMat);
    hipFree(d_destImage);
    exit(0);
}
